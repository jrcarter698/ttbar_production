#include "hip/hip_runtime.h"
#define SECDEC_RESULT_IS_COMPLEX 1
#include "common_cuda.h"

#define SecDecInternalSignCheckErrorPositivePolynomial(id) {val = nan("U"); break;}
#define SecDecInternalSignCheckErrorContourDeformation(id) {val = nan("F"); break;}

extern "C" __global__ void
A1111_integral__sector_6_order_0(
    result_t * __restrict__ result,
    const uint64_t lattice,
    const uint64_t index1,
    const uint64_t index2,
    const uint64_t * __restrict__ genvec,
    const real_t * __restrict__ shift,
    const real_t * __restrict__ realp,
    const complex_t * __restrict__ complexp,
    const real_t * __restrict__ deformp
)
{
    // assert(blockDim.x == 128);
    const uint64_t bid = blockIdx.x;
    const uint64_t tid = threadIdx.x;
    const real_t s = realp[0]; (void)s;
    const real_t t = realp[1]; (void)t;
    const real_t m2 = realp[2]; (void)m2;
    const real_t SecDecInternalLambda0 = deformp[0];
    const real_t SecDecInternalLambda1 = deformp[1];
    const real_t SecDecInternalLambda2 = deformp[2];
    const real_t invlattice = 1.0/lattice;
    result_t val = 0.0;
    uint64_t index = index1 + (bid*128 + tid)*8;
    uint64_t li_x0 = mulmod(index, genvec[0], lattice);
    uint64_t li_x1 = mulmod(index, genvec[1], lattice);
    uint64_t li_x2 = mulmod(index, genvec[2], lattice);
    for (uint64_t i = 0; (i < 8) && (index < index2); i++, index++) {
        real_t x0 = warponce(li_x0*invlattice + shift[0], 1.0);
        li_x0 = warponce_i(li_x0 + genvec[0], lattice);
        real_t x1 = warponce(li_x1*invlattice + shift[1], 1.0);
        li_x1 = warponce_i(li_x1 + genvec[1], lattice);
        real_t x2 = warponce(li_x2*invlattice + shift[2], 1.0);
        li_x2 = warponce_i(li_x2 + genvec[2], lattice);
        real_t w_x0 = korobov3x3_w(x0);
        real_t w_x1 = korobov3x3_w(x1);
        real_t w_x2 = korobov3x3_w(x2);
        real_t w = w_x0*w_x1*w_x2;
        x0 = clamp01(korobov3x3_f(x0));
        x1 = clamp01(korobov3x3_f(x1));
        x2 = clamp01(korobov3x3_f(x2));
        auto tmp1_1 = 2*m2;
        auto tmp1_2 = 4*x0;
        auto tmp1_3 = x2*m2;
        auto tmp3_1 = tmp1_3*tmp1_2;
        auto tmp1_4 = tmp1_1*x1;
        auto tmp1_5 = tmp1_4 + m2;
        auto tmp1_6 = x0*m2;
        auto tmp3_2 = tmp1_6 + tmp1_3;
        auto tmp3_3 = 2*tmp3_2;
        auto tmp1_7 = x1*m2;
        auto tmp1_8 = tmp1_7*x2;
        auto tmp1_9 = 4*tmp1_7;
        auto tmp3_4 = x0*tmp1_9;
        auto tmp3_5 = tmp3_4 + m2+4*tmp1_8;
        auto tmp3_6 = m2 + tmp1_7;
        auto tmp1_10 = x2*tmp1_4;
        auto tmp3_7 = tmp1_10 + m2;
        auto tmp1_11 = x0*tmp1_4;
        auto tmp3_8 = tmp1_11 + tmp3_7;
        auto tmp3_9 = 2*tmp3_8;
        auto tmp1_12 = 2*x0;
        auto tmp3_10 = tmp3_7*tmp1_12;
        auto tmp3_11 = tmp1_3 + tmp3_10;
        auto tmp1_13 = x0*tmp1_5;
        auto tmp3_12 = tmp1_13 + tmp1_8 + m2 + tmp1_3;
        auto tmp3_13 = x0*tmp1_1*x2;
        auto tmp3_14 = -1 + x2;
        auto tmp3_15 = SecDecInternalLambda2*tmp3_14;
        auto tmp1_14 = -1+2*x2;
        auto tmp3_16 = SecDecInternalLambda2*tmp1_14;
        auto tmp1_15 = SecDecInternalLambda2*x2;
        auto tmp1_16 = SecDecInternalLambda1*x1;
        auto tmp1_17 = -1+2*x1;
        auto tmp3_17 = SecDecInternalLambda1*tmp1_17;
        auto tmp1_18 = SecDecInternalLambda0*x0;
        auto tmp3_18 = -1 + tmp1_12;
        auto tmp3_19 = SecDecInternalLambda0*tmp3_18;
        auto __PowCall1 = x0*x0;
        auto __PowCall2 = x1*x1;
        auto __PowCall3 = x2*x2;
        auto tmp2_10 = __PowCall1 + __PowCall3;
        auto tmp2_11 = tmp1_4*tmp2_10;
        auto tmp3_20 = tmp2_11 + tmp3_11;
        auto tmp2_12 = m2*tmp2_10;
        auto tmp3_21 = tmp3_13 + tmp2_12;
        auto tmp3_22 = __PowCall2*tmp3_21;
        auto tmp3_23 = tmp3_12 + tmp3_22;
        auto tmp2_13 = tmp3_3*__PowCall2;
        auto tmp2_14 = tmp2_13 + tmp3_6;
        auto tmp3_24 = tmp2_13 + tmp1_5;
        auto tmp3_25 = tmp1_1*tmp2_10;
        auto tmp3_26 = tmp3_1 + tmp3_25;
        auto tmp2_15 = tmp1_1*__PowCall2;
        auto __RealPartCall1 = SecDecInternalRealPart(tmp2_15);
        auto __RealPartCall3 = SecDecInternalRealPart(tmp3_9);
        auto __RealPartCall5 = SecDecInternalRealPart(tmp3_5);
        auto tmp3_27 = SecDecInternalLambda0*__PowCall1;
        auto tmp3_28 = tmp3_27-tmp1_18;
        auto tmp3_29 = SecDecInternalI(__RealPartCall3);
        auto tmp3_30 = tmp3_29*tmp3_28;
        auto tmp3_31 = SecDecInternalI(__RealPartCall1);
        auto tmp3_32 = tmp3_31*tmp3_28;
        auto tmp3_33 = SecDecInternalLambda1*__PowCall2;
        auto tmp3_34 = tmp3_33-tmp1_16;
        auto tmp3_35 = tmp3_29*tmp3_34;
        auto tmp3_36 = SecDecInternalI(__RealPartCall5);
        auto tmp3_37 = tmp3_36*tmp3_34;
        auto tmp3_38 = SecDecInternalLambda2*__PowCall3;
        auto tmp3_39 = tmp3_38-tmp1_15;
        auto tmp3_40 = tmp3_31*tmp3_39;
        auto tmp3_41 = tmp3_36*tmp3_39;
        auto __RealPartCall2 = SecDecInternalRealPart(tmp3_20);
        auto __RealPartCall4 = SecDecInternalRealPart(tmp2_14);
        auto __RealPartCall6 = SecDecInternalRealPart(tmp3_24);
        auto __RealPartCall7 = SecDecInternalRealPart(tmp3_26);
        auto tmp3_42 = SecDecInternalI(__RealPartCall4);
        auto tmp3_43 = tmp3_15*tmp3_42;
        auto tmp3_44 = 1 + tmp3_43;
        auto tmp3_45 = SecDecInternalLambda0*__PowCall1;
        auto tmp3_46 = tmp3_45-tmp1_18;
        auto tmp3_47 = SecDecInternalI(__RealPartCall6);
        auto tmp3_48 = tmp3_47*tmp3_46;
        auto tmp3_49 = x0 + tmp3_48;
        auto tmp3_50 = SecDecInternalI(__RealPartCall1);
        auto tmp3_51 = tmp3_50*tmp3_46;
        auto tmp3_52 = tmp3_19*tmp3_47;
        auto tmp3_53 = tmp3_52+1 + tmp3_51;
        auto tmp3_54 = SecDecInternalLambda1*__PowCall2;
        auto tmp3_55 = tmp3_54-tmp1_16;
        auto tmp3_56 = SecDecInternalI(__RealPartCall2);
        auto tmp3_57 = tmp3_56*tmp3_55;
        auto tmp3_58 = x1 + tmp3_57;
        auto tmp3_59 = SecDecInternalI(__RealPartCall7*tmp3_55);
        auto tmp3_60 = tmp3_17*tmp3_56;
        auto tmp3_61 = tmp3_60+1 + tmp3_59;
        auto tmp3_62 = SecDecInternalLambda2*__PowCall3;
        auto tmp3_63 = tmp3_62-tmp1_15;
        auto tmp2_16 = tmp3_42*tmp3_63;
        auto tmp3_64 = x2 + tmp2_16;
        auto tmp3_65 = tmp3_50*tmp3_63;
        auto tmp3_66 = tmp3_16*tmp3_42;
        auto tmp3_67 = tmp3_66+1 + tmp3_65;
        auto tmp3_68 = -tmp3_61*tmp3_40;
        auto tmp3_69 = tmp3_35*tmp3_41;
        auto tmp3_70 = tmp3_68 + tmp3_69;
        auto tmp3_71 = tmp3_32*tmp3_70;
        auto tmp3_72 = tmp3_37*tmp3_40;
        auto tmp3_73 = -tmp3_35*tmp3_67;
        auto tmp3_74 = tmp3_72 + tmp3_73;
        auto tmp3_75 = tmp3_30*tmp3_74;
        auto tmp3_76 = -tmp3_37*tmp3_41;
        auto tmp3_77 = tmp3_61*tmp3_67;
        auto tmp3_78 = tmp3_76 + tmp3_77;
        auto tmp3_79 = tmp3_53*tmp3_78;
        auto tmp3_80 = tmp3_79 + tmp3_71 + tmp3_75;
        auto __PowCall4 = tmp3_49*tmp3_49;
        auto __PowCall5 = tmp3_58*tmp3_58;
        auto __PowCall6 = tmp3_64*tmp3_64;
        auto tmp3_81 = tmp3_58+1;
        auto tmp3_82 = tmp3_64*tmp3_81;
        auto tmp3_83 = __PowCall4 + __PowCall6;
        auto tmp3_84 = __PowCall5*tmp3_83;
        auto tmp3_85 = tmp3_84 + tmp3_82+1 + tmp3_49;
        auto tmp3_86 = m2*tmp3_85;
        auto tmp3_87 = __PowCall5*tmp3_64;
        auto tmp3_88 = tmp3_58 + tmp3_87;
        auto tmp3_89 = tmp3_88*tmp3_49*tmp1_1;
        auto tmp3_90 = tmp3_86 + tmp3_89;
        auto tmp3_91 = tmp3_49 + tmp3_64;
        auto tmp3_92 = __PowCall5*tmp3_91;
        auto tmp3_93 = tmp3_92 + tmp3_81;
        auto __PowCall7 = tmp3_90*tmp3_90*tmp3_90;
        auto __DenominatorCall1 = SecDecInternalDenominator(__PowCall7);
        auto tmp3_94 = -tmp3_23 + tmp3_90;
        auto tmp3_95 = x2*tmp3_80*tmp3_44*__DenominatorCall1*tmp3_93;
        auto _SignCheckExpression = SecDecInternalImagPart(tmp3_94);
        if (unlikely(_SignCheckExpression>0)) SecDecInternalSignCheckErrorContourDeformation(1);
        auto tmp3_96 = SecDecInternalRealPart(tmp3_93);
        if (unlikely(tmp3_96<0)) SecDecInternalSignCheckErrorPositivePolynomial(1);
        val += w*(tmp3_95);
    }
    // Sum up 128*8=1024 values across 4 warps.
    typedef hipcub::BlockReduce<result_t, 128, hipcub::BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY> Reduce;
    __shared__ typename Reduce::TempStorage shared;
    result_t sum = Reduce(shared).Sum(val);
    if (tid == 0) result[bid] = sum;
}
