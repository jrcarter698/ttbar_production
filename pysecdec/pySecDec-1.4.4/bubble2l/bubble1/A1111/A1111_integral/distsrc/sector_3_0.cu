#include "hip/hip_runtime.h"
#define SECDEC_RESULT_IS_COMPLEX 1
#include "common_cuda.h"

#define SecDecInternalSignCheckErrorPositivePolynomial(id) {val = nan("U"); break;}
#define SecDecInternalSignCheckErrorContourDeformation(id) {val = nan("F"); break;}

extern "C" __global__ void
A1111_integral__sector_3_order_0(
    result_t * __restrict__ result,
    const uint64_t lattice,
    const uint64_t index1,
    const uint64_t index2,
    const uint64_t * __restrict__ genvec,
    const real_t * __restrict__ shift,
    const real_t * __restrict__ realp,
    const complex_t * __restrict__ complexp,
    const real_t * __restrict__ deformp
)
{
    // assert(blockDim.x == 128);
    const uint64_t bid = blockIdx.x;
    const uint64_t tid = threadIdx.x;
    const real_t s = realp[0]; (void)s;
    const real_t t = realp[1]; (void)t;
    const real_t m2 = realp[2]; (void)m2;
    const real_t SecDecInternalLambda0 = deformp[0];
    const real_t SecDecInternalLambda1 = deformp[1];
    const real_t SecDecInternalLambda2 = deformp[2];
    const real_t invlattice = 1.0/lattice;
    result_t val = 0.0;
    uint64_t index = index1 + (bid*128 + tid)*8;
    uint64_t li_x0 = mulmod(index, genvec[0], lattice);
    uint64_t li_x1 = mulmod(index, genvec[1], lattice);
    uint64_t li_x2 = mulmod(index, genvec[2], lattice);
    for (uint64_t i = 0; (i < 8) && (index < index2); i++, index++) {
        real_t x0 = warponce(li_x0*invlattice + shift[0], 1.0);
        li_x0 = warponce_i(li_x0 + genvec[0], lattice);
        real_t x1 = warponce(li_x1*invlattice + shift[1], 1.0);
        li_x1 = warponce_i(li_x1 + genvec[1], lattice);
        real_t x2 = warponce(li_x2*invlattice + shift[2], 1.0);
        li_x2 = warponce_i(li_x2 + genvec[2], lattice);
        real_t w_x0 = korobov3x3_w(x0);
        real_t w_x1 = korobov3x3_w(x1);
        real_t w_x2 = korobov3x3_w(x2);
        real_t w = w_x0*w_x1*w_x2;
        x0 = clamp01(korobov3x3_f(x0));
        x1 = clamp01(korobov3x3_f(x1));
        x2 = clamp01(korobov3x3_f(x2));
        auto tmp1_1 = 2*m2;
        auto tmp1_2 = 1 + x0;
        auto tmp3_1 = 2*tmp1_2 + x2;
        auto tmp3_2 = m2*tmp3_1;
        auto tmp1_3 = x1 + x0+2;
        auto tmp1_4 = tmp1_3+2*x2;
        auto tmp3_3 = m2*tmp1_4;
        auto tmp1_5 = 1 + x2;
        auto tmp3_4 = m2*tmp1_5;
        auto tmp3_5 = x2*tmp1_3;
        auto tmp3_6 = tmp3_5+1 + x1+2*x0;
        auto tmp3_7 = m2*tmp3_6;
        auto tmp1_6 = x2*SecDecInternalLambda2;
        auto tmp1_7 = -SecDecInternalLambda2 + tmp1_6;
        auto tmp1_8 = -SecDecInternalLambda2+2*tmp1_6;
        auto tmp1_9 = x1*SecDecInternalLambda1;
        auto tmp1_10 = -SecDecInternalLambda1+2*tmp1_9;
        auto tmp1_11 = x0*SecDecInternalLambda0;
        auto tmp1_12 = -SecDecInternalLambda0+2*tmp1_11;
        auto __PowCall1 = x0*x0;
        auto __PowCall2 = x1*x1;
        auto __PowCall3 = x2*x2;
        auto __RealPartCall1 = SecDecInternalRealPart(tmp1_1);
        auto __RealPartCall2 = SecDecInternalRealPart(m2);
        auto tmp2_5 = __PowCall3 + __PowCall1;
        auto tmp3_8 = m2*tmp2_5;
        auto tmp3_9 = tmp3_8 + tmp3_7;
        auto tmp2_6 = SecDecInternalLambda0*__PowCall1;
        auto tmp3_10 = -tmp1_11 + tmp2_6;
        auto tmp2_7 = SecDecInternalI(__RealPartCall2);
        auto tmp3_11 = tmp2_7*tmp3_10;
        auto tmp2_8 = SecDecInternalLambda1*__PowCall2;
        auto tmp3_12 = -tmp1_9 + tmp2_8;
        auto tmp3_13 = tmp2_7*tmp3_12;
        auto tmp2_9 = SecDecInternalLambda2*__PowCall3;
        auto tmp3_14 = tmp2_9-tmp1_6;
        auto tmp3_15 = tmp3_14*tmp2_7;
        auto __RealPartCall3 = SecDecInternalRealPart(tmp3_4);
        auto __RealPartCall4 = SecDecInternalRealPart(tmp3_3);
        auto __RealPartCall5 = SecDecInternalRealPart(tmp3_2);
        auto tmp3_16 = SecDecInternalI(__RealPartCall4);
        auto tmp3_17 = tmp1_7*tmp3_16;
        auto tmp3_18 = 1 + tmp3_17;
        auto tmp2_10 = SecDecInternalLambda0*__PowCall1;
        auto tmp3_19 = tmp2_10-tmp1_11;
        auto tmp2_11 = SecDecInternalI(__RealPartCall5);
        auto tmp2_12 = tmp2_11*tmp3_19;
        auto tmp3_20 = x0 + tmp2_12;
        auto tmp2_13 = SecDecInternalI(__RealPartCall1);
        auto tmp3_21 = tmp2_13*tmp3_19;
        auto tmp3_22 = tmp1_12*tmp2_11;
        auto tmp3_23 = tmp3_22+1 + tmp3_21;
        auto tmp3_24 = SecDecInternalLambda1*__PowCall2;
        auto tmp3_25 = -tmp1_9 + tmp3_24;
        auto tmp2_14 = SecDecInternalI(__RealPartCall3);
        auto tmp3_26 = tmp2_14*tmp3_25;
        auto tmp3_27 = x1 + tmp3_26;
        auto tmp3_28 = tmp1_10*tmp2_14;
        auto tmp3_29 = 1 + tmp3_28;
        auto tmp2_15 = SecDecInternalLambda2*__PowCall3;
        auto tmp3_30 = tmp2_15-tmp1_6;
        auto tmp2_16 = tmp3_16*tmp3_30;
        auto tmp3_31 = x2 + tmp2_16;
        auto tmp3_32 = tmp2_13*tmp3_30;
        auto tmp3_33 = tmp1_8*tmp3_16;
        auto tmp3_34 = tmp3_33+1 + tmp3_32;
        auto tmp3_35 = tmp3_27 + tmp3_20+1 + tmp3_31;
        auto tmp3_36 = tmp3_34*tmp3_29;
        auto tmp3_37 = -tmp3_13*tmp3_15;
        auto tmp3_38 = tmp3_36 + tmp3_37;
        auto tmp3_39 = tmp3_23*tmp3_38;
        auto tmp3_40 = -tmp3_11*tmp3_15*tmp3_29;
        auto tmp3_41 = tmp3_40 + tmp3_39;
        auto __PowCall4 = tmp3_20*tmp3_20;
        auto __PowCall5 = tmp3_31*tmp3_31;
        auto tmp3_42 = tmp3_20 + tmp3_31;
        auto tmp3_43 = tmp1_1*tmp3_42;
        auto tmp3_44 = tmp3_20 + tmp3_27;
        auto tmp3_45 = tmp3_31*tmp3_44;
        auto tmp3_46 = tmp3_45 + tmp3_27 + __PowCall5+1 + __PowCall4;
        auto tmp3_47 = m2*tmp3_46;
        auto tmp3_48 = tmp3_47 + tmp3_43;
        auto __PowCall6 = tmp3_48*tmp3_48*tmp3_48;
        auto __DenominatorCall1 = SecDecInternalDenominator(__PowCall6);
        auto tmp3_49 = -tmp3_9 + tmp3_48;
        auto tmp3_50 = x2*tmp3_41*tmp3_18*__DenominatorCall1*tmp3_35;
        auto _SignCheckExpression = SecDecInternalImagPart(tmp3_49);
        if (unlikely(_SignCheckExpression>0)) SecDecInternalSignCheckErrorContourDeformation(1);
        auto tmp3_51 = SecDecInternalRealPart(tmp3_35);
        if (unlikely(tmp3_51<0)) SecDecInternalSignCheckErrorPositivePolynomial(1);
        val += w*(tmp3_50);
    }
    // Sum up 128*8=1024 values across 4 warps.
    typedef hipcub::BlockReduce<result_t, 128, hipcub::BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY> Reduce;
    __shared__ typename Reduce::TempStorage shared;
    result_t sum = Reduce(shared).Sum(val);
    if (tid == 0) result[bid] = sum;
}
