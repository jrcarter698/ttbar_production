#include "hip/hip_runtime.h"
#define SECDEC_RESULT_IS_COMPLEX 1
#include "common_cuda.h"

#define SecDecInternalSignCheckErrorPositivePolynomial(id) {val = nan("U"); break;}
#define SecDecInternalSignCheckErrorContourDeformation(id) {val = nan("F"); break;}

extern "C" __global__ void
A111m3_integral__sector_1_order_1(
    result_t * __restrict__ result,
    const uint64_t lattice,
    const uint64_t index1,
    const uint64_t index2,
    const uint64_t * __restrict__ genvec,
    const real_t * __restrict__ shift,
    const real_t * __restrict__ realp,
    const complex_t * __restrict__ complexp,
    const real_t * __restrict__ deformp
)
{
    // assert(blockDim.x == 128);
    const uint64_t bid = blockIdx.x;
    const uint64_t tid = threadIdx.x;
    const real_t s = realp[0]; (void)s;
    const real_t t = realp[1]; (void)t;
    const real_t m2 = realp[2]; (void)m2;
    const real_t SecDecInternalLambda0 = deformp[0];
    const real_t SecDecInternalLambda1 = deformp[1];
    const real_t invlattice = 1.0/lattice;
    result_t val = 0.0;
    uint64_t index = index1 + (bid*128 + tid)*8;
    uint64_t li_x0 = mulmod(index, genvec[0], lattice);
    uint64_t li_x1 = mulmod(index, genvec[1], lattice);
    for (uint64_t i = 0; (i < 8) && (index < index2); i++, index++) {
        real_t x0 = warponce(li_x0*invlattice + shift[0], 1.0);
        li_x0 = warponce_i(li_x0 + genvec[0], lattice);
        real_t x1 = warponce(li_x1*invlattice + shift[1], 1.0);
        li_x1 = warponce_i(li_x1 + genvec[1], lattice);
        real_t w_x0 = korobov3x3_w(x0);
        real_t w_x1 = korobov3x3_w(x1);
        real_t w = w_x0*w_x1;
        x0 = clamp01(korobov3x3_f(x0));
        x1 = clamp01(korobov3x3_f(x1));
        auto tmp1_1 = 2*m2;
        auto tmp1_2 = x1+1;
        auto tmp1_3 = x0 + tmp1_2;
        auto tmp3_1 = tmp1_3*tmp1_1;
        auto tmp1_4 = 1+2*x0;
        auto tmp1_5 = tmp1_4+2*x1;
        auto tmp3_2 = m2*tmp1_5;
        auto tmp3_3 = m2*tmp1_4*tmp1_2;
        auto tmp3_4 = 1.68E+2*m2;
        auto tmp1_6 = 1.2E+2*m2;
        auto tmp1_7 = x1*SecDecInternalLambda1;
        auto tmp1_8 = -SecDecInternalLambda1+2*tmp1_7;
        auto tmp1_9 = x0*SecDecInternalLambda0;
        auto tmp1_10 = -SecDecInternalLambda0+2*tmp1_9;
        auto __PowCall1 = x0*x0;
        auto __PowCall2 = x1*x1;
        auto __PowCall3 = m2*m2;
        auto __PowCall4 = m2*m2*m2;
        auto __RealPartCall1 = SecDecInternalRealPart(tmp1_1);
        auto tmp2_4 = __PowCall2 + __PowCall1;
        auto tmp3_5 = m2*tmp2_4;
        auto tmp3_6 = tmp3_5 + tmp3_3;
        auto tmp2_5 = SecDecInternalLambda0*__PowCall1;
        auto tmp3_7 = -tmp1_9 + tmp2_5;
        auto tmp2_6 = SecDecInternalI(__RealPartCall1);
        auto tmp3_8 = tmp2_6*tmp3_7;
        auto tmp2_7 = SecDecInternalLambda1*__PowCall2;
        auto tmp3_9 = -tmp1_7 + tmp2_7;
        auto tmp3_10 = tmp2_6*tmp3_9;
        auto __RealPartCall2 = SecDecInternalRealPart(tmp3_2);
        auto __RealPartCall3 = SecDecInternalRealPart(tmp3_1);
        auto tmp3_11 = SecDecInternalLambda0*__PowCall1;
        auto tmp3_12 = tmp3_11-tmp1_9;
        auto tmp3_13 = SecDecInternalI(__RealPartCall3);
        auto tmp3_14 = tmp3_13*tmp3_12;
        auto tmp3_15 = x0 + tmp3_14;
        auto tmp2_8 = SecDecInternalI(__RealPartCall1);
        auto tmp3_16 = tmp2_8*tmp3_12;
        auto tmp3_17 = tmp1_10*tmp3_13;
        auto tmp3_18 = tmp3_17+1 + tmp3_16;
        auto tmp3_19 = SecDecInternalLambda1*__PowCall2;
        auto tmp3_20 = tmp3_19-tmp1_7;
        auto tmp2_9 = SecDecInternalI(__RealPartCall2);
        auto tmp2_10 = tmp2_9*tmp3_20;
        auto tmp3_21 = x1 + tmp2_10;
        auto tmp3_22 = tmp2_8*tmp3_20;
        auto tmp3_23 = tmp1_8*tmp2_9;
        auto tmp3_24 = tmp3_23+1 + tmp3_22;
        auto tmp3_25 = tmp3_21+1 + tmp3_15;
        auto tmp3_26 = -tmp3_8*tmp3_10;
        auto tmp3_27 = tmp3_18*tmp3_24;
        auto tmp3_28 = tmp3_26 + tmp3_27;
        auto __PowCall5 = tmp3_15*tmp3_15;
        auto __PowCall6 = tmp3_15*tmp3_15*tmp3_15;
        auto __PowCall7 = tmp3_21*tmp3_21;
        auto tmp3_29 = tmp3_21+1;
        auto tmp3_30 = __PowCall5 + __PowCall7 + tmp3_29;
        auto tmp3_31 = m2*tmp3_30;
        auto tmp3_32 = tmp3_15*tmp1_1*tmp3_29;
        auto tmp3_33 = tmp3_32 + tmp3_31;
        auto _logCall2 = SecDecInternalLog(tmp3_25);
        auto __PowCall10 = tmp3_25*tmp3_25;
        auto __PowCall11 = tmp3_25*tmp3_25*tmp3_25;
        auto _logCall1 = SecDecInternalLog(tmp3_33);
        auto __PowCall8 = tmp3_33*tmp3_33;
        auto __PowCall9 = tmp3_33*tmp3_33*tmp3_33;
        auto __PowCall12 = __PowCall10*__PowCall10;
        auto __DenominatorCall1 = SecDecInternalDenominator(tmp3_33);
        auto tmp3_34 = 2*tmp3_15 + __PowCall5+1;
        auto tmp3_35 = tmp3_34*tmp3_33*__PowCall10*__PowCall3;
        auto tmp3_36 = tmp3_15+1;
        auto tmp3_37 = tmp3_36*__PowCall8*tmp3_25;
        auto tmp3_38 = tmp1_6*tmp3_37;
        auto tmp3_39 = -2.4E+1*tmp3_35-1.2E+2*__PowCall9 + tmp3_38;
        auto tmp3_40 = -tmp3_4*tmp3_37;
        auto tmp3_41 = tmp3_15 + __PowCall5;
        auto tmp3_42 = -1-__PowCall6-3*tmp3_41;
        auto tmp3_43 = __PowCall11*__PowCall4*tmp3_42;
        auto tmp3_44 = 4.8E+1*tmp3_35+2*tmp3_43+1.48E+2*__PowCall9 + tmp3_40;
        auto tmp3_45 = __PowCall11*__PowCall12;
        auto __DenominatorCall2 = SecDecInternalDenominator(tmp3_45);
        auto tmp3_46 = -tmp3_6 + tmp3_33;
        auto tmp3_47 = 2*_logCall2-_logCall1;
        auto tmp3_48 = tmp3_47*tmp3_39;
        auto tmp3_49 = tmp3_44 + tmp3_48;
        auto tmp3_50 = tmp3_28*__DenominatorCall1*__DenominatorCall2*tmp3_49;
        auto _SignCheckExpression = SecDecInternalImagPart(tmp3_46);
        if (unlikely(_SignCheckExpression>0)) SecDecInternalSignCheckErrorContourDeformation(1);
        auto tmp3_51 = SecDecInternalRealPart(tmp3_25);
        if (unlikely(tmp3_51<0)) SecDecInternalSignCheckErrorPositivePolynomial(1);
        val += w*(tmp3_50);
    }
    // Sum up 128*8=1024 values across 4 warps.
    typedef hipcub::BlockReduce<result_t, 128, hipcub::BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY> Reduce;
    __shared__ typename Reduce::TempStorage shared;
    result_t sum = Reduce(shared).Sum(val);
    if (tid == 0) result[bid] = sum;
}
